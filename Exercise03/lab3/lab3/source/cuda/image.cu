#include "hip/hip_runtime.h"
#include "image.cuh"

#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

__global__ void grayscale_kernel(const Pixel<std::uint8_t>* const input, Pixel<std::uint8_t>* const output, const unsigned int width, const unsigned int height) {
    //Calculate the global threadID
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        unsigned int index = y * width + x;
        Pixel<std::uint8_t> pixel = input[index];
        Pixel<std::uint8_t>* pixelPtr = &pixel;
        auto* r = reinterpret_cast<std::uint8_t*>(pixelPtr);
        auto* g = reinterpret_cast<std::uint8_t*>(pixelPtr + 1);
        auto* b = reinterpret_cast<std::uint8_t*>(pixelPtr + 2);

        // Calculate the grayscale value
        const auto gray = *r * 0.2989 + *g * 0.5870 + *b * 0.1140;
        auto gray_converted = static_cast<std::uint8_t>(gray);

        output[index] = reinterpret_cast<Pixel<std::uint8_t>&>(gray_converted, gray_converted, gray_converted);
    }
}

BitmapImage get_grayscale_cuda(const BitmapImage& source) {
    const size_t num_pixels = source.get_width() * source.get_height();
    const auto raw_size = num_pixels * sizeof(Pixel<std::uint8_t>);
    Pixel<std::uint8_t>* output;
    Pixel<std::uint8_t>* device_input;
    Pixel<std::uint8_t>* device_output;

    // Allocate memory on the device
    hipMalloc((void**)&device_input, raw_size);
    hipMalloc((void**)&device_output, raw_size);

    // Copy input data to the device
    hipMemcpy(device_input, source.get_data(), raw_size, hipMemcpyHostToDevice);

    dim3 thread_dim(32, 32);
    dim3 block_dim(divup(source.get_width(), thread_dim.x), divup(source.get_height(), thread_dim.y));

    // Call the kernel
    grayscale_kernel<<<block_dim, thread_dim>>>(device_input, device_output, source.get_width(), source.get_height());

    // Copy the result back to the host
    output = new Pixel<std::uint8_t>[num_pixels];
    hipMemcpy(output, device_output, raw_size, hipMemcpyDeviceToHost);


    // Free the device memory
    hipFree(device_input);
    hipFree(device_output);

    // Wait for the kernel to finish
    hipDeviceSynchronize();

    BitmapImage result{source.get_width(), source.get_height()};
    
    for (unsigned int x = 0; x < source.get_width(); x++) {
        for (unsigned int y = 0; y < source.get_height(); y++) {
            const auto pixel = output[y * source.get_width() + x];
            result.set_pixel(x, y, pixel);
        }
    }


    // Free the host memory
    delete[] output;

    return result;
}