#include "hip/hip_runtime.h"
#include "image.cuh"

#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

//TODO kompiliert nocht nicht vollständig Error:"calling a host function("Pixel<unsigned char> ::get_red_channel() const") from a global function("grayscale_kernel") is not allowed"
__global__ void grayscale_kernel(const Pixel<std::uint8_t>* const input, Pixel<std::uint8_t>* const output, const unsigned int width, const unsigned int height) {
    //Calculate the global threadID
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        unsigned int index = y * width + x;
        Pixel<std::uint8_t> pixel = input[index];
        const auto r = pixel.get_red_channel();
        const auto g = pixel.get_green_channel();
        const auto b = pixel.get_blue_channel();


        // Calculate the grayscale value
        const auto gray = r * 0.2989 + g * 0.5870 + b * 0.1140;
        const auto gray_converted = static_cast<std::uint8_t>(gray);

        const auto gray_pixel = Pixel{ gray_converted , gray_converted,  gray_converted };

        output[index] = gray_pixel;
    }
}

BitmapImage get_grayscale_cuda(const BitmapImage& source) {
    const size_t num_pixels = source.get_width() * source.get_height();
    Pixel<std::uint8_t>* output;
    Pixel<std::uint8_t>* device_input;
    Pixel<std::uint8_t>* device_output;

    // Allocate memory on the device
    hipMalloc((void**)&device_input, num_pixels * sizeof(Pixel<std::uint8_t>));
    hipMalloc((void**)&device_output, num_pixels * sizeof(Pixel<std::uint8_t>));

    // Copy input data to the device
    hipMemcpy(device_input, source.get_data(), num_pixels * sizeof(Pixel<std::uint8_t>), hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim(divup(source.get_width(), blockDim.x), divup(source.get_height(), blockDim.y));

    // Call the kernel
    grayscale_kernel<<<gridDim, blockDim>>>(device_input, device_output, source.get_width(), source.get_height());

    // Wait for the kernel to finish
    hipDeviceSynchronize();

    // Copy the result back to the host
    output = new Pixel<std::uint8_t>[num_pixels];
    hipMemcpy(output, device_output, num_pixels * sizeof(Pixel<std::uint8_t>), hipMemcpyDeviceToHost);

    BitmapImage result{source.get_width(), source.get_height()};
    
    for (unsigned int x = 0; x < source.get_width(); x++) {
        for (unsigned int y = 0; y < source.get_height(); y++) {
            const auto pixel = output[y * source.get_width() + x];
            result.set_pixel(x, y, pixel);
        }
    }

    // Free the device memory
    hipFree(device_input);
    hipFree(device_output);

    // Free the host memory
    delete[] output;

    return result;
}