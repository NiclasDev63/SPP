#include "hip/hip_runtime.h"
#include "image.cuh"

#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

//TODO kompiliert nocht nicht vollständig
__global__ void grayscale_kernel(const Pixel<std::uint8_t>* const input, Pixel<std::uint8_t>* const output, const unsigned int width, const unsigned int height) {
    //Calculate the global threadID
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        unsigned int index = y * width + x;
        Pixel<std::uint8_t> pixel = input[index];
        const auto r = pixel.get_red_channel();
        const auto g = pixel.get_green_channel();
        const auto b = pixel.get_blue_channel();

        // Calculate the grayscale value
        const auto gray = r * 0.2989 + g * 0.5870 + b * 0.1140;
        const auto gray_converted = static_cast<std::uint8_t>(gray);

        const auto gray_pixel = Pixel{ gray_converted , gray_converted,  gray_converted };

        output[index] = gray_pixel;
    }
}