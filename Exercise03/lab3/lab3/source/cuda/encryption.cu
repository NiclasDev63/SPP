#include "hip/hip_runtime.h"
#include "encryption.cuh"

#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

__global__ void hash(const std::uint64_t* const values, std::uint64_t* const hashes, const unsigned int length) {
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < length) {
		constexpr auto val_a = std::uint64_t{ 5'647'095'006'226'412'969 };
		constexpr auto val_b = std::uint64_t{ 41'413'938'183'913'153 };
		constexpr auto val_c = std::uint64_t{ 6'225'658'194'131'981'369 };

		const std::uint64_t value = values[index];

		const auto val_1 = (value >> 14) + val_a;
		const auto val_2 = (value << 54) ^ val_b;
		const auto val_3 = (val_1 + val_2) << 4;
		const auto val_4 = (value % val_c) * 137;

		const auto final_hash = val_3 ^ val_4;

		hashes[index] = final_hash;
	}
}

__global__ void flat_hash(const std::uint64_t* const values, std::uint64_t* const hashes, const unsigned int length) {
	//Determine how many values each thread has to hash. Considering there is only one {tx, 1, 1} block
	int max = blockDim.x;
	int values_per_thread = (length + max - 1) / max;
	
	unsigned int index = threadIdx.x * values_per_thread;

	if (index < length) {

		constexpr auto val_a = std::uint64_t{ 5'647'095'006'226'412'969 };
		constexpr auto val_b = std::uint64_t{ 41'413'938'183'913'153 };
		constexpr auto val_c = std::uint64_t{ 6'225'658'194'131'981'369 };

		for (int i = 0; i < values_per_thread; i++) {
			if (index + i < length) {
				const auto value = values[index + i];

				const auto val_1 = (value >> 14) + val_a;
				const auto val_2 = (value << 54) ^ val_b;
				const auto val_3 = (val_1 + val_2) << 4;
				const auto val_4 = (value % val_c) * 137;

				hashes[index + i] = val_3 ^ val_4;
			}
		}
	}
}

__global__ void find_hash(const std::uint64_t* const hashes, unsigned int* const indices, const unsigned int length, const std::uint64_t searched_hash, unsigned int* const ptr) {
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < length) {
		//Mark if thread at index contains matching hash.
		if (hashes[index] == searched_hash) {
			indices[index] = 1;
		}
	}
}

__global__ void hash_schemes(std::uint64_t* const hashes, const unsigned int length) {
	//Determine how many schemes each thread has to hash.
	int max = blockDim.x * blockIdx.x;
	int values_per_thread = (length + max - 1) / max;

	unsigned int index = (blockIdx.x * blockDim.x + threadIdx.x) * values_per_thread;

	if (index < length) {
			constexpr auto val_a = std::uint64_t{ 5'647'095'006'226'412'969 };
			constexpr auto val_b = std::uint64_t{ 41'413'938'183'913'153 };
			constexpr auto val_c = std::uint64_t{ 6'225'658'194'131'981'369 };

		for (int i = 0; i < values_per_thread; i++) {
			if (index + i < length) {
				//Calculate encoded scheme from index
				std::uint64_t scheme = index + i;
				scheme |= (scheme << 32);

				//Hash scheme and write to hashes[]
				const auto val_1 = (scheme >> 14) + val_a;
				const auto val_2 = (scheme << 54) ^ val_b;
				const auto val_3 = (val_1 + val_2) << 4;
				const auto val_4 = (scheme % val_c) * 137;

				hashes[index + i] = val_3 ^ val_4;
			}
		}	
	}
}